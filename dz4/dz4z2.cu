#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define ACCURACY 0.01

#define MAXLINE 128
#define PIXPERLINE 16

char c[MAXLINE];

void pgmsize(char* filename, int* nx, int* ny);
void pgmread(char* filename, void* vp, int nxmax, int nymax, int* nx, int* ny);
void pgmwrite(char* filename, void* vx, int nx, int ny);

double** dosharpen(char* infile, int nx, int ny);
//double** dosharpenParallel(char* infile, int nx, int ny);
double filter(int d, int i, int j);

int** int2Dmalloc(int nx, int ny);
double** double2Dmalloc(int nx, int ny);

//void compareSharp(int w, int h, double** sequential, double** parallel);

double wtime();
void pgmsize(char* filename, int* nx, int* ny)
{
	FILE* fp;

	if (NULL == (fp = fopen(filename, "r")))
	{
		fprintf(stderr, "pgmsize: cannot open <%s>\n", filename);
		exit(-1);
	}

	fgets(c, MAXLINE, fp);
	fgets(c, MAXLINE, fp);

	fscanf(fp, "%d %d", nx, ny);

	fclose(fp);
}

void pgmread(char* filename, void* vp, int nxmax, int nymax, int* nx, int* ny)
{
	FILE* fp;

	int nxt, nyt, i, j, t;

	int* pixmap = (int*)vp;

	if (NULL == (fp = fopen(filename, "r")))
	{
		fprintf(stderr, "pgmread: cannot open <%s>\n", filename);
		exit(-1);
	}

	fgets(c, MAXLINE, fp);
	fgets(c, MAXLINE, fp);

	fscanf(fp, "%d %d", nx, ny);

	nxt = *nx;
	nyt = *ny;

	if (nxt > nxmax || nyt > nymax)
	{
		fprintf(stderr, "pgmread: image larger than array\n");
		fprintf(stderr, "nxmax, nymax, nxt, nyt = %d, %d, %d, %d\n",
			nxmax, nymax, nxt, nyt);
		exit(-1);
	}

	fscanf(fp, "%d", &t);

	for (j = 0; j < nyt; j++)
	{
		for (i = 0; i < nxt; i++)
		{
			fscanf(fp, "%d", &t);
			pixmap[(nyt - j - 1) + nyt * i] = t;
		}
	}

	fclose(fp);
}

void pgmwrite(char* filename, void* vx, int nx, int ny)
{
	FILE* fp;

	int i, j, k, grey;

	double xmin, xmax, tmp;
	double thresh = 255.0;

	double* x = (double*)vx;

	if (NULL == (fp = fopen(filename, "w")))
	{
		fprintf(stderr, "pgmwrite: cannot create <%s>\n", filename);
		exit(-1);
	}

	xmin = fabs(x[0]);
	xmax = fabs(x[0]);

	for (i = 0; i < nx * ny; i++)
	{
		if (fabs(x[i]) < xmin)
			xmin = fabs(x[i]);
		if (fabs(x[i]) > xmax)
			xmax = fabs(x[i]);
	}

	fprintf(fp, "P2\n");
	fprintf(fp, "# Written by pgmwrite\n");
	fprintf(fp, "%d %d\n", nx, ny);
	fprintf(fp, "%d\n", (int)thresh);

	k = 0;

	for (j = ny - 1; j >= 0; j--)
	{
		for (i = 0; i < nx; i++)
		{

			tmp = x[j + ny * i];

			if (xmin < 0 || xmax > thresh)
			{
				tmp = (int)((thresh * ((fabs(tmp - xmin)) / (xmax - xmin))) + 0.5);
			}
			else
			{
				tmp = (int)(fabs(tmp) + 0.5);
			}

			grey = tmp;

			fprintf(fp, "%3d ", grey);

			if (0 == (k + 1) % PIXPERLINE)
				fprintf(fp, "\n");

			k++;
		}
	}

	if (0 != k % PIXPERLINE)
		fprintf(fp, "\n");
	fclose(fp);
}

double wtime(void)
{
	return 0;
}

double filter(int d, int i, int j)
{
	double rd4sq, rsq, sigmad4sq, sigmasq, x, y, delta;

	int d4 = 4;

	double sigmad4 = 1.4;
	double filter0 = -40.0;

	rd4sq = d4 * d4;
	rsq = d * d;

	sigmad4sq = sigmad4 * sigmad4;
	sigmasq = sigmad4sq * (rsq / rd4sq);

	x = (double)i;
	y = (double)j;

	rsq = x * x + y * y;

	delta = rsq / (2.0 * sigmasq);

	return (filter0 * (1.0 - delta) * exp(-delta));
}

int** int2Dmalloc(int nx, int ny)
{
	int i;
	int** idata;

	idata = (int**)malloc(nx * sizeof(int*) + nx * ny * sizeof(int));

	idata[0] = (int*)(idata + nx);

	for (i = 1; i < nx; i++)
	{
		idata[i] = idata[i - 1] + ny;
	}

	return idata;
}

double** double2Dmalloc(int nx, int ny)
{
	int i;
	double** ddata;

	ddata = (double**)malloc(nx * sizeof(double*) + nx * ny * sizeof(double));

	ddata[0] = (double*)(ddata + nx);

	for (i = 1; i < nx; i++)
	{
		ddata[i] = ddata[i - 1] + ny;
	}

	return ddata;
}

double** dosharpen(char* infile, int nx, int ny)
{
	int d = 8;

	double norm = (2 * d - 1) * (2 * d - 1);
	double scale = 2.0;

	int xpix, ypix, pixcount;

	int i, j, k, l;
	double tstart, tstop, time;

	int** fuzzy = int2Dmalloc(nx, ny);								/* Will store the fuzzy input image when it is first read in from file */
	double** fuzzyPadded = double2Dmalloc(nx + 2 * d, ny + 2 * d);  /* Will store the fuzzy input image plus additional border padding */
	double** convolutionPartial = double2Dmalloc(nx, ny);			/* Will store the convolution of the filter with parts of the fuzzy image computed by individual processes */
	double** convolution = double2Dmalloc(nx, ny);					/* Will store the convolution of the filter with the full fuzzy image */
	double** sharp = double2Dmalloc(nx, ny);						/* Will store the sharpened image obtained by adding rescaled convolution to the fuzzy image */
	double** sharpCropped = double2Dmalloc(nx - 2 * d, ny - 2 * d); /* Will store the sharpened image cropped to remove a border layer distorted by the algorithm */

	char outfile[256];
	strcpy(outfile, infile);
	*(strchr(outfile, '.')) = '\0';
	strcat(outfile, "_sharpened.pgm");

	for (i = 0; i < nx; i++)
	{
		for (j = 0; j < ny; j++)
		{
			fuzzy[i][j] = 0;
			sharp[i][j] = 0.0;
		}
	}

	// printf("Using a filter of size %d x %d\n", 2 * d + 1, 2 * d + 1);
	// printf("\n");

	// printf("Reading image file: %s\n", infile);
	// fflush(stdout);

	pgmread(infile, &fuzzy[0][0], nx, ny, &xpix, &ypix);

	// printf("... done\n\n");
	// fflush(stdout);

	if (xpix == 0 || ypix == 0 || nx != xpix || ny != ypix)
	{
		printf("Error reading %s\n", infile);
		fflush(stdout);
		exit(-1);
	}

	for (i = 0; i < nx + 2 * d; i++)
	{
		for (j = 0; j < ny + 2 * d; j++)
		{
			fuzzyPadded[i][j] = 0.0;
		}
	}

	for (i = 0; i < nx; i++)
	{
		for (j = 0; j < ny; j++)
		{
			fuzzyPadded[i + d][j + d] = fuzzy[i][j];
		}
	}

	// printf("Starting calculation ...\n");

	// fflush(stdout);

	tstart = wtime();

	pixcount = 0;

	for (i = 0; i < nx; i++)
	{
		for (j = 0; j < ny; j++)
		{
			for (k = -d; k <= d; k++)
			{
				for (l = -d; l <= d; l++)
				{
					convolution[i][j] = convolution[i][j] + filter(d, k, l) * fuzzyPadded[i + d + k][j + d + l];
				}
			}
			pixcount += 1;
		}
	}

	tstop = wtime();
	time = tstop - tstart;

	// printf("... finished\n");
	// printf("\n");
	// fflush(stdout);

	for (i = 0; i < nx; i++)
	{
		for (j = 0; j < ny; j++)
		{
			sharp[i][j] = fuzzyPadded[i + d][j + d] - scale / norm * convolution[i][j];
		}
	}

	// printf("Writing output file: %s\n", outfile);
	// printf("\n");

	for (i = d; i < nx - d; i++)
	{
		for (j = d; j < ny - d; j++)
		{
			sharpCropped[i - d][j - d] = sharp[i][j];
		}
	}

	pgmwrite(outfile, &sharpCropped[0][0], nx - 2 * d, ny - 2 * d);

	// printf("... done\n");
	// printf("\n");
	// printf("Calculation time was %f seconds\n", time);
	// fflush(stdout);

	free(fuzzy);
	free(fuzzyPadded);
	free(convolutionPartial);
	free(convolution);
	// free(sharp);
	free(sharpCropped);

	return sharp;
}

double** makeFilterMatrix(int d)
{
    double** matrix = (double**)malloc((2 * d + 1) * sizeof(double*));

    for (int i = 0; i <= 2 * d; i++)
    {
        matrix[i] = (double*)malloc((2 * d + 1) * sizeof(double));

        for (int j = -d; j <= d; j++)
            matrix[i][j + d] = filter(d, i - d, j);
    }

    return matrix;
}

#define TILE_SIZE 16

__global__
void sharpenKernel(double* filterMatrix, double* fuzzyPadded, double* convolution, int nx, int ny)
{
    // __shared__ double sharedFuzzyPadded[32][32];
    /*int misses = 0;
    for (int k = -d; k <= d; k++)
        for (int l = -d; l <= d; l++) {
            double t1 = dev_FilterMatrix[(k + d) * 17 + l + d];
            double t2 = filter(8, k, l);

            if (t1 != t2)
                misses++;

        }*/

    const int d = 8;
    const double norm = (2 * d - 1) * (2 * d - 1);
    int i = blockIdx.y * TILE_SIZE + threadIdx.y;
    int j = blockIdx.x * TILE_SIZE + threadIdx.x;

    /*if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        for (int k = -d; k <= d; k++)
            for (int l = -d; l <= d; l++)
                sharedFuzzyPadded[i + d + k][j + d + l] = fuzzyPadded[i + d + k][j + d + l];
    }
    __syncthreads();*/

    if (i < nx && j < ny) {
        //sharp[i][j] = 0.0;

        for (int k = -d; k <= d; k++)
        {
            for (int l = -d; l <= d; l++)
            {
                convolution[i * ny + j] = convolution[i * ny + j] + filterMatrix[(k + d) * (17) + l + d] * fuzzyPadded[(i + d + k) * (ny + 2 * d) + j + d + l];
            }
        }

        convolution[i * ny + j] *= (2.0 / norm);
    }
}

double** sharpen_cuda_init(char* infile, int nx, int ny, float* ms)
{
    const int d = 8;
    //////////////////////////////////////
    // LUT

    double* dev_FilterMatrix;



    double** filterMatrix = makeFilterMatrix(d);
    size_t filterMatrixSize = (2 * d + 1) * (2 * d + 1) * sizeof(double);
    hipMalloc(&dev_FilterMatrix, filterMatrixSize);
    hipMemcpy(dev_FilterMatrix, &filterMatrix[0][0], filterMatrixSize, hipMemcpyHostToDevice);
    //hipMemcpyToSymbol(HIP_SYMBOL(dev_FilterMatrix), &filterMatrix[0][0], filterMatrixSize);
    //////////////////////////////////////

    double** sharp = double2Dmalloc(nx, ny);
    double** sharpCropped = double2Dmalloc(nx - 2 * d, ny - 2 * d);
    int** fuzzy = int2Dmalloc(nx, ny);
    double** fuzzyPadded = double2Dmalloc(nx + 2 * d, ny + 2 * d);
    pgmread(infile, &fuzzy[0][0], nx, ny, &nx, &ny);

    for (int i = 0; i < nx; i++)
    {
        for (int j = 0; j < ny; j++)
        {
            fuzzy[i][j] = 0;
            sharp[i][j] = 0.0;
        }
    }

    for (int i = 0; i < nx + 2 * d; i++)
    {
        for (int j = 0; j < ny + 2 * d; j++)
        {
            fuzzyPadded[i][j] = 0.0;
        }
    }

    for (int i = 0; i < nx; i++)
        for (int j = 0; j < ny; j++)
            fuzzyPadded[i + d][j + d] = fuzzy[i][j];

    double* devFuzzyPadded;
    size_t devFuzzyPadded_size = (nx + 2 * d) * (ny + 2 * d) * sizeof(double);

    hipMalloc(&devFuzzyPadded, devFuzzyPadded_size);
    hipMemcpy(devFuzzyPadded, &fuzzyPadded[0][0], devFuzzyPadded_size, hipMemcpyHostToDevice);

    //////////////////////////////////////

    //double** devSharpCropped;
    //size_t sharpCroppedSize = (nx - 2 * d) * (ny - 2 * d) * sizeof(double);
    //hipMalloc(&devSharpCropped, sharpCroppedSize);
    //////////////////////////////////////

    double** convolution = double2Dmalloc(nx, ny);

    size_t convolutionSize = (nx) * (ny) * sizeof(double);
    size_t sharpSize = (nx) * (ny) * sizeof(double);
    double* devConvolution;
    hipMalloc(&devConvolution, convolutionSize);

    int gridX_size = ceil(nx / TILE_SIZE);
    int gridY_size = ceil(ny / TILE_SIZE);

    dim3 dimGrid(gridX_size, gridY_size);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    sharpenKernel << <dimGrid, dimBlock >> > (dev_FilterMatrix, devFuzzyPadded, devConvolution, nx, ny);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(ms, start, stop);


    hipMemcpy(&convolution[0][0], devConvolution, convolutionSize, hipMemcpyDeviceToHost);


    for (int i = 0; i < nx; i++)
        for (int j = 0; j < ny; j++)
            sharp[i][j] = fuzzyPadded[i + d][j + d] - convolution[i][j];

    for (int i = d; i < nx - d; i++)
        for (int j = d; j < ny - d; j++)
            sharpCropped[i - d][j - d] = sharp[i][j];

    hipFree(dev_FilterMatrix);
    hipFree(devFuzzyPadded);
    //hipFree(devSharpCropped);
    hipFree(devConvolution);
    //hipFree(devSharp);

    free(fuzzy);
    free(fuzzyPadded);
    free(convolution);
    free(sharp);

    return sharpCropped;
}

void compareSharp(int w, int h, double** sequential, double** parallel)
{
    int misses = 0;

    for (int i = 0; i < w; i++)
    {
        for (int j = 0; j < h; j++)
        {
            if (fabs(sequential[i][j] - parallel[i][j]) > ACCURACY)
                misses++;
        }
    }

    if (misses > ACCURACY * (w * h))
        printf("Test FAILED\n");
    else
        printf("Test PASSED\n");
}

int main(int argc, char* argv[])
{
	double tstart, tstop, time, timeParallel;

	char* filename;
	int xpix, ypix;

	if (argc < 2)
		return 1;

	filename = argv[1];

	// printf("\n");
	// printf("Image sharpening code running in serial\n");
	// printf("\n");
	// printf("Input file is: %s\n", filename);

	pgmsize(filename, &xpix, &ypix);

	// printf("Image size is %d x %d\n", xpix, ypix);
	// printf("\n");

	tstart = wtime();

	double** sharpSequential = dosharpen(filename, xpix, ypix);

	tstop = wtime();
	time = tstop - tstart;

  tstart = wtime();
  
  float ms;
	double** sharpParallel = sharpen_cuda_init(filename, xpix, ypix, &ms);

	tstop = wtime();
	timeParallel = tstop - tstart;

	printf("Input file: %s\n", filename);
	printf("Sequential execution time: %f\n", time);
	printf("Parallel execution time: %f\n", timeParallel);

	compareSharp(xpix, ypix, sharpSequential, sharpParallel);

	printf("\n");

	free(sharpSequential);
	//free(sharpParallel);

	return 0;
}
