#include "hip/hip_runtime.h"
# include <math.h>
# include <stdio.h>
# include <stdlib.h>
# include <string.h>
# include <time.h>

#include "hip/hip_runtime.h"
#include ""

# define DEFAULT_H 500
# define DEFAULT_W 500
# define DEFAULT_CNT 200
# define DEFAULT_FILENAME "julia"

int main(int argc, char* argv[]);
unsigned char* julia_set(int w, int h, int cnt, float xl, float xr, float yb, float yt);
unsigned char* julia_set_parallel(int w, int h, int cnt, float xl, float xr, float yb, float yt);
int julia(int w, int h, float xl, float xr, float yb, float yt, int i, int j, int cnt);
void julia_parallel(int w, int h, float xl, float xr, float yb, float yt, int i, int j, int cnt, int* val);
void tga_write(int w, int h, unsigned char rgb[], char* filename);
void timestamp();

#define ACCURACY 0.01

void tga_compare(int w, int h, unsigned char* rgb_sequential, unsigned char* rgb_parallel)
{
	char failed = 0;

	for (int i = 0; i < 3 * w * h; i++)
	{
		if (fabs(rgb_sequential[i] - rgb_parallel[i]) > ACCURACY)
		{
			failed = 1;
			break;
		}
	}

	if (failed)
		printf("Test FAILED\n");
	else
		printf("Test PASSED\n");
}

int main(int argc, char* argv[]) {
	int h = DEFAULT_H;
	int w = DEFAULT_W;
	int cnt = DEFAULT_CNT;
	char filename[256] = DEFAULT_FILENAME;
	char buffer[256];
	unsigned char* rgb;
	unsigned char* rgbParallel;
	float xl = -1.5;
	float xr = +1.5;
	float yb = -1.5;
	float yt = +1.5;

	if (argc == 4) {
		h = atoi(argv[1]);
		w = atoi(argv[2]);
		cnt = atoi(argv[3]);
		if (!h || !w || !cnt) return 1;
	}

	strcat(filename, "_");
	sprintf(buffer, "%d", h);
	strcat(filename, buffer);
	strcat(filename, "_");
	sprintf(buffer, "%d", w);
	strcat(filename, buffer);
	strcat(filename, "_");
	sprintf(buffer, "%d", cnt);
	strcat(filename, buffer);
	strcat(filename, ".tga");

	clock_t timeSequential = clock();
	rgb = julia_set(w, h, cnt, xl, xr, yb, yt);
	timeSequential = clock() - timeSequential;

	double t_seq = (double)(timeSequential) / CLOCKS_PER_SEC;
	printf("\tSequential execution time: %f\n", t_seq);

	rgbParallel = julia_set_parallel(w, h, cnt, xl, xr, yb, yt);

	tga_compare(w, h, rgb, rgbParallel);

	tga_write(w, h, rgb, "sequential.tga");
	tga_write(w, h, rgbParallel, filename);

	free(rgb);
	free(rgbParallel);

	return 0;
}

unsigned char* julia_set(int w, int h, int cnt, float xl, float xr, float yb, float yt)
{
	int i;
	int j;
	int juliaValue;
	int k;
	unsigned char* rgb;

	rgb = (unsigned char*)malloc(w * h * 3 * sizeof(unsigned char));

	for (j = 0; j < h; j++)
	{
		for (i = 0; i < w; i++)
		{
			juliaValue = julia(w, h, xl, xr, yb, yt, i, j, cnt);

			k = 3 * (j * w + i);

			rgb[k] = 255 * (1 - juliaValue);
			rgb[k + 1] = 255 * (1 - juliaValue);
			rgb[k + 2] = 255;
		}
	}


	return rgb;
}

int julia(int w, int h, float xl, float xr, float yb, float yt, int i, int j, int cnt)
{
	float ai;
	float ar;
	float ci = 0.156;
	float cr = -0.8;
	int k;
	float t;
	float x;
	float y;

	x = ((float)(w - i - 1) * xl
		+ (float)(i)*xr)
		/ (float)(w - 1);

	y = ((float)(h - j - 1) * yb
		+ (float)(j)*yt)
		/ (float)(h - 1);

	ar = x;
	ai = y;

	for (k = 0; k < cnt; k++)
	{
		t = ar * ar - ai * ai + cr;
		ai = ar * ai + ai * ar + ci;
		ar = t;

		if (1000 < ar * ar + ai * ai)
		{
			return 0;
		}
	}

	return 1;
}

#define TILE_WIDTH 16

__global__
void juliaValueKernel(void* rgb_void, int w, int h, int cnt, float xl, float xr, float yb, float yt, int WIDTH, int HEIGHT)
{
	int i = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int j = blockIdx.x * TILE_WIDTH + threadIdx.x;

	if (i < w && j < h) {
		unsigned char* rgb = (unsigned char*)rgb_void;

		int juliaValue;
		julia_parallel(w, h, xl, xr, yb, yt, i, j, cnt, &juliaValue);

		int k = 3 * (j * w + i);

		rgb[k] = 255 * (1 - juliaValue);
		rgb[k + 1] = 255 * (1 - juliaValue);
		rgb[k + 2] = 255;
	}
}

unsigned char* julia_set_parallel(int w, int h, int cnt, float xl, float xr, float yb, float yt)
{
	unsigned char* rgb;
	void* dev_rgb;

	size_t size_rgb = w * h * 3 * sizeof(unsigned char);

	rgb = (unsigned char*)malloc(size_rgb);
	hipMalloc(&dev_rgb, size_rgb);

	/*const int numOfElements = w * h;
	const int tileArea = TILE_WIDTH * TILE_WIDTH;

	int numOfBlocks = numOfElements / tileArea;
	if (numOfElements % tileArea)
		numOfBlocks++;*/
	double tx = ceil((double)w / TILE_WIDTH);
	double ty = ceil((double)h / TILE_WIDTH);
	printf("%f %f\n", tx, ty);

	dim3 gridSize((int)tx, (int)ty);
	dim3 blockSize(TILE_WIDTH, TILE_WIDTH);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	juliaValueKernel << < gridSize, blockSize >> > ((void*)dev_rgb, w, h, cnt, xl, xr, yb, yt, w, h);
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms, start, stop);
	printf("\tParallel execution time: %f\n", ms);

	hipMemcpy(rgb, dev_rgb, size_rgb, hipMemcpyDeviceToHost);
	hipFree(dev_rgb);

	return rgb;
}

__device__
void julia_parallel(int w, int h, float xl, float xr, float yb, float yt, int i, int j, int cnt, int* val)
{
	float ai;
	float ar;
	float ci = 0.156;
	float cr = -0.8;
	int k;
	float t;
	float x;
	float y;

	x = ((float)(w - i - 1) * xl
		+ (float)(i)*xr)
		/ (float)(w - 1);

	y = ((float)(h - j - 1) * yb
		+ (float)(j)*yt)
		/ (float)(h - 1);

	ar = x;
	ai = y;

	int returnValue = 1;

	for (k = 0; k < cnt; k++)
	{
		t = ar * ar - ai * ai + cr;
		ai = ar * ai + ai * ar + ci;
		ar = t;

		if (1000 < ar * ar + ai * ai)
		{
			returnValue = 0;
			break;
		}
	}

	*val = returnValue;
}

void tga_write(int w, int h, unsigned char rgb[], char* filename)
{
	FILE* file_unit;
	unsigned char header1[12] = { 0,0,2,0,0,0,0,0,0,0,0,0 };
	unsigned char header2[6] = { w % 256, w / 256, h % 256, h / 256, 24, 0 };

	file_unit = fopen(filename, "wb");

	fwrite(header1, sizeof(unsigned char), 12, file_unit);
	fwrite(header2, sizeof(unsigned char), 6, file_unit);

	fwrite(rgb, sizeof(unsigned char), 3 * w * h, file_unit);

	fclose(file_unit);

	printf("--> Graphics data saved as '%s'\n", filename);

	return;
}

void timestamp(void)
{
# define TIME_SIZE 40

	static char time_buffer[TIME_SIZE];
	const struct tm* tm;
	time_t now;

	now = time(NULL);
	tm = localtime(&now);

	strftime(time_buffer, TIME_SIZE, "%d %B %Y %I:%M:%S %p", tm);

	printf("%s\n", time_buffer);

	return;
# undef TIME_SIZE
}
